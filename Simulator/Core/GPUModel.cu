#include "hip/hip_runtime.h"
/**
 * @file GPUModel.cu
 * 
 * @ingroup Simulator/Core
 *
 * @brief Implementation of Model for the spiking neural networks.
 * 
 */

#include "GPUModel.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Connections.h"
#include "Global.h"
#include "AllVertices.h"
#include "AllEdges.h"

#ifdef PERFORMANCE_METRICS
float g_time;
hipEvent_t start, stop;
#endif // PERFORMANCE_METRICS

__constant__ int d_debug_mask[1];

GPUModel::GPUModel() :
  Model::Model(),
  edgeIndexMapDevice_(nullptr),
  randNoise_d(nullptr),
  allVerticesDevice_(nullptr),
  allEdgesDevice_(nullptr)
{
}

GPUModel::~GPUModel() 
{
  //Let Model base class handle de-allocation
}

/// Allocates  and initializes memories on CUDA device.
/// @param[out] allVerticesDevice          Memory location of the pointer to the vertices list on device memory.
/// @param[out] allEdgesDevice         Memory location of the pointer to the edges list on device memory.
void GPUModel::allocDeviceStruct(void** allVerticesDevice, void** allEdgesDevice)
{
  // Get vertices and edges
  shared_ptr<AllVertices> vertices = layout_->getVertices();
  shared_ptr<AllEdges> edges = connections_->getEdges();

  // Allocate Vertices and Edges structs on GPU device memory
  vertices->allocNeuronDeviceStruct(allVerticesDevice);
  edges->allocEdgeDeviceStruct(allEdgesDevice);

  // Allocate memory for random noise array
  int numVertices = Simulator::getInstance().getTotalVertices();
  BGSIZE randNoise_d_size = numVertices * sizeof (float);	// size of random noise array
  HANDLE_ERROR( hipMalloc ( ( void ** ) &randNoise_d, randNoise_d_size ) );

  // Copy host vertex and edge arrays into GPU device
  vertices->copyNeuronHostToDevice( *allVerticesDevice );
  edges->copyEdgeHostToDevice( *allEdgesDevice );

  // Allocate edge inverse map in device memory
  allocSynapseImap( numVertices );
}

/// Copies device memories to host memories and deallocates them.
/// @param[out] allVerticesDevice          Memory location of the pointer to the vertices list on device memory.
/// @param[out] allEdgesDevice         Memory location of the pointer to the edges list on device memory.
void GPUModel::deleteDeviceStruct(void** allVerticesDevice, void** allEdgesDevice)
{  
  // Get vertices and edges
  shared_ptr<AllVertices> vertices = layout_->getVertices();
  shared_ptr<AllEdges> edges = connections_->getEdges();

  // Copy device edge and vertex structs to host memory
  vertices->copyNeuronDeviceToHost( *allVerticesDevice);
  // Deallocate device memory
  vertices->deleteNeuronDeviceStruct( *allVerticesDevice);
  // Copy device edge and vertex structs to host memory
  edges->copyEdgeDeviceToHost( *allEdgesDevice);
  // Deallocate device memory
  edges->deleteEdgeDeviceStruct( *allEdgesDevice );
  HANDLE_ERROR( hipFree( randNoise_d ) );
}

/// Sets up the Simulation.
void GPUModel::setupSim()
{
  // Set device ID
  HANDLE_ERROR( hipSetDevice( g_deviceId ) );
  // Set DEBUG flag
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(d_debug_mask), &g_debug_mask, sizeof(int) ) );
  Model::setupSim();

  //initialize Mersenne Twister
  //assuming numVertices >= 100 and is a multiple of 100. Note rng_mt_rng_count must be <= MT_RNG_COUNT
  int rng_blocks = 25; //# of blocks the kernel will use
  int rng_nPerRng = 4; //# of iterations per thread (thread granularity, # of rands generated per thread)
  int rng_mt_rng_count = Simulator::getInstance().getTotalVertices() / rng_nPerRng; //# of threads to generate for numVertices rand #s
  int rng_threads = rng_mt_rng_count/rng_blocks; //# threads per block needed
  initMTGPU(Simulator::getInstance().getNoiseRngSeed(), rng_blocks, rng_threads, rng_nPerRng, rng_mt_rng_count);

#ifdef PERFORMANCE_METRICS
  hipEventCreate( &start );
  hipEventCreate( &stop );

  t_gpu_rndGeneration = 0.0;
  t_gpu_advanceNeurons = 0.0;
  t_gpu_advanceSynapses = 0.0;
  t_gpu_calcSummation = 0.0;
#endif // PERFORMANCE_METRICS

  // allocates memories on CUDA device
  allocDeviceStruct((void **)&allVerticesDevice_, (void **)&allEdgesDevice_);

  // copy inverse map to the device memory
  copyEdgeIndexMapHostToDevice(*(connections_->getEdgeIndexMap().get()), Simulator::getInstance().getTotalVertices());

  // set some parameters used for advanceVerticesDevice
  layout_->getVertices()->setAdvanceVerticesDeviceParams(*(connections_->getEdges().get()));

  // set some parameters used for advanceEdgesDevice
  connections_->getEdges()->setAdvanceEdgesDeviceParams();
}

/// Performs any finalization tasks on network following a simulation.
void GPUModel::finish()
{
  // deallocates memories on CUDA device
  deleteDeviceStruct((void**)&allVerticesDevice_, (void**)&allEdgesDevice_);
  deleteSynapseImap();

#ifdef PERFORMANCE_METRICS
  hipEventDestroy( start );
  hipEventDestroy( stop );
#endif // PERFORMANCE_METRICS
}

/// Advance everything in the model one time step. In this case, that
/// means calling all of the kernels that do the "micro step" updating
/// (i.e., NOT the stuff associated with growth).
void GPUModel::advance()
{
#ifdef PERFORMANCE_METRICS
  // Reset CUDA timer to start measurement of GPU operations
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // Get vertices and edges
  shared_ptr<AllVertices> vertices = layout_->getVertices();
  shared_ptr<AllEdges> edges = connections_->getEdges();

  normalMTGPU(randNoise_d);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_rndGeneration);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // display running info to console
  // Advance vertices ------------->
   dynamic_cast<AllSpikingNeurons *>(vertices.get())->advanceVertices(*(connections_->getEdges().get()), allVerticesDevice_, allEdgesDevice_, randNoise_d, edgeIndexMapDevice_);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_advanceNeurons);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // Advance edges ------------->
  edges->advanceEdges(allEdgesDevice_, allVerticesDevice_, edgeIndexMapDevice_);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_advanceSynapses);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // calculate summation point
  calcSummationMap();

#ifdef PERFORMANCE_METRICS
 cudaLapTime(t_gpu_calcSummation);
#endif // PERFORMANCE_METRICS
}

/// Add psr of all incoming edges to summation points.
void GPUModel::calcSummationMap()
{
  // CUDA parameters
  const int threadsPerBlock = 256;
  int blocksPerGrid = ( Simulator::getInstance().getTotalVertices() + threadsPerBlock - 1 ) / threadsPerBlock;

  calcSummationMapDevice <<< blocksPerGrid, threadsPerBlock >>> (
        Simulator::getInstance().getTotalVertices(), allVerticesDevice_, edgeIndexMapDevice_, allEdgesDevice_ );
}

/// Update the connection of all the Vertices and Edges of the simulation.
void GPUModel::updateConnections()
{
  // Get vertices and edges
  shared_ptr<AllVertices> vertices = layout_->getVertices();
  shared_ptr<AllEdges> edges = connections_->getEdges();

  dynamic_cast<AllSpikingNeurons*>(vertices.get())->copyNeuronDeviceSpikeCountsToHost(allVerticesDevice_);
  dynamic_cast<AllSpikingNeurons*>(vertices.get())->copyNeuronDeviceSpikeHistoryToHost(allVerticesDevice_);

  // Update Connections data
  if (connections_->updateConnections(*(vertices.get()), layout_.get())) {
    connections_->updateEdgesWeights(Simulator::getInstance().getTotalVertices(), *(vertices.get()), *(edges.get()), allVerticesDevice_, allEdgesDevice_, layout_.get());
    // create edge index map
    connections_->createEdgeIndexMap();
    // copy index map to the device memory
    copyEdgeIndexMapHostToDevice(*(connections_->getEdgeIndexMap().get()), Simulator::getInstance().getTotalVertices());
  }
}

/// Update the Neuron's history.
void GPUModel::updateHistory()
{
  Model::updateHistory();
  // clear spike count
  
  shared_ptr<AllVertices> vertices = layout_->getVertices();
  dynamic_cast<AllSpikingNeurons*>(vertices.get())->clearNeuronSpikeCounts(allVerticesDevice_);
}

/// Allocate device memory for edge inverse map.
/// @param  count	The number of vertices.
void GPUModel::allocSynapseImap( int count )
{
  EdgeIndexMap synapseIMapDevice;

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.outgoingEdgeBegin_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.outgoingEdgeCount_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.outgoingEdgeBegin_, 0, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.outgoingEdgeCount_, 0, count * sizeof( BGSIZE ) ) );

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.incomingEdgeBegin_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.incomingEdgeCount_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.incomingEdgeBegin_, 0, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.incomingEdgeCount_, 0, count * sizeof( BGSIZE ) ) );

  HANDLE_ERROR( hipMalloc( ( void ** ) &edgeIndexMapDevice_, sizeof( EdgeIndexMap ) ) );
  HANDLE_ERROR( hipMemcpy( edgeIndexMapDevice_, &synapseIMapDevice, sizeof( EdgeIndexMap ), 
        hipMemcpyHostToDevice ) );
}

/// Deallocate device memory for edge inverse map.
void GPUModel::deleteSynapseImap(  )
{
  EdgeIndexMap synapseIMapDevice;

  HANDLE_ERROR( hipMemcpy ( &synapseIMapDevice, edgeIndexMapDevice_, 
        sizeof( EdgeIndexMap ), hipMemcpyDeviceToHost ) );

  HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingEdgeBegin_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingEdgeCount_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingEdgeIndexMap_ ) );

  HANDLE_ERROR( hipFree( synapseIMapDevice.incomingEdgeBegin_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.incomingEdgeCount_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.incomingEdgeIndexMap_ ) );

  HANDLE_ERROR( hipFree( edgeIndexMapDevice_ ) );
}

/// Copy EdgeIndexMap in host memory to EdgeIndexMap in device memory.
/// @param  synapseIndexMapHost		Reference to the EdgeIndexMap in host memory.
void GPUModel::copyEdgeIndexMapHostToDevice(EdgeIndexMap &synapseIndexMapHost, int numVertices)
{
  shared_ptr<AllEdges> edges = connections_->getEdges();
  int totalSynapseCount = dynamic_cast<AllEdges*>(edges.get())->totalEdgeCount_;

  if (totalSynapseCount == 0)
    return;

  // TODO: rename variable, DevicePointer
  EdgeIndexMap synapseIMapDevice;

  HANDLE_ERROR( hipMemcpy ( &synapseIMapDevice, edgeIndexMapDevice_, 
        sizeof( EdgeIndexMap ), hipMemcpyDeviceToHost ) );

  // forward map
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.outgoingEdgeBegin_, 
        synapseIndexMapHost.outgoingEdgeBegin_, numVertices * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.outgoingEdgeCount_, 
        synapseIndexMapHost.outgoingEdgeCount_, numVertices * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  // the number of edges may change, so we reallocate the memory
  if (synapseIMapDevice.outgoingEdgeIndexMap_ != nullptr) {
    HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingEdgeIndexMap_ ) );
  }
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.outgoingEdgeIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.outgoingEdgeIndexMap_, synapseIndexMapHost.outgoingEdgeIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );

  // active edge map
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.incomingEdgeBegin_, 
        synapseIndexMapHost.incomingEdgeBegin_, numVertices * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.incomingEdgeCount_, 
        synapseIndexMapHost.incomingEdgeCount_, numVertices * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  // the number of edges may change, so we reallocate the memory
  if (synapseIMapDevice.incomingEdgeIndexMap_ != nullptr) {
    HANDLE_ERROR( hipFree( synapseIMapDevice.incomingEdgeIndexMap_ ) );
  }
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.incomingEdgeIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.incomingEdgeIndexMap_, synapseIndexMapHost.incomingEdgeIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );

  HANDLE_ERROR( hipMemcpy ( edgeIndexMapDevice_, &synapseIMapDevice, 
        sizeof( EdgeIndexMap ), hipMemcpyHostToDevice ) );
}

/// Calculate the sum of synaptic input to each vertex.
///
/// Calculate the sum of synaptic input to each vertex. One thread
/// corresponds to one vertex. Iterates sequentially through the
/// forward edge index map (edgeIndexMapDevice_) to access only
/// existing edges. Using this structure eliminates the need to skip
/// edges that have undergone lazy deletion from the main
/// (allEdgesDevice) edge structure. The forward map is
/// re-computed during each network restructure (once per epoch) to
/// ensure that all edge pointers for a vertex are stored
/// contiguously.
/// 
/// @param[in] totalVertices           Number of vertices in the entire simulation.
/// @param[in,out] allVerticesDevice   Pointer to Neuron structures in device memory.
/// @param[in] edgeIndexMapDevice_  Pointer to forward map structures in device memory.
/// @param[in] allEdgesDevice      Pointer to Edge structures in device memory.
__global__ void calcSummationMapDevice(int totalVertices, 
				       AllSpikingNeuronsDeviceProperties* __restrict__ allVerticesDevice, 
				       const EdgeIndexMap* __restrict__ edgeIndexMapDevice_, 
				       const AllSpikingSynapsesDeviceProperties* __restrict__ allEdgesDevice)
{
  // The usual thread ID calculation and guard against excess threads
  // (beyond the number of vertices, in this case).
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= totalVertices )
    return;

  // Number of incoming edges
  const BGSIZE synCount = edgeIndexMapDevice_->incomingEdgeCount_[idx];
  // Optimization: terminate thread if no incoming edges
  if (synCount != 0) {
    // Index of start of this vertex's block of forward map entries
    const int beginIndex = edgeIndexMapDevice_->incomingEdgeBegin_[idx];
    // Address of the start of this vertex's block of forward map entries
    const BGSIZE* activeMapBegin = 
      &(edgeIndexMapDevice_->incomingEdgeIndexMap_[beginIndex]);
    // Summed post-synaptic response (PSR)
    BGFLOAT sum = 0.0;
    // Index of the current incoming edge
    BGSIZE synIndex;
    // Repeat for each incoming edge
    for (BGSIZE i = 0; i < synCount; i++) {
      // Get index of current incoming edge
      synIndex = activeMapBegin[i];
      // Fetch its PSR and add into sum
      sum += allEdgesDevice->psr_[synIndex];
    }
    // Store summed PSR into this vertex's summation point
    allVerticesDevice->summationMap_[idx] = sum;
  }
}

/// Copy GPU Edge data to CPU.
void GPUModel::copyGPUtoCPU()
{
  // copy device edge structs to host memory
  connections_->getEdges()->copyEdgeDeviceToHost(allEdgesDevice_);
}

/// Copy CPU Edge data to GPU.
void GPUModel::copyCPUtoGPU()
{
  // copy host edge structs to device memory
  connections_->getEdges()->copyEdgeHostToDevice(allEdgesDevice_);
}

/// Print out SynapseProps on the GPU.
void GPUModel::printGPUSynapsesPropsModel() const
{  
  connections_->getEdges()->printGPUEdgesProps(allEdgesDevice_);
}

